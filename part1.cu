
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 512
#endif

using namespace std;

const int FILTER_WIDTH = 3;

//We will only use this filter in part 1
int FILTER[FILTER_WIDTH*FILTER_WIDTH] = {
    0, -1, 0, 
    -1, 5, -1, 
    0, -1, 0
};

// Display the first and last 10 items
// For debug only
void displayResult(const int original[], const int result[], int size) {
	cout << "Display result: ";

	for (int i = 0; i < 10; i++) {
		cout << result[i] << "\n";
	}
	cout << ".\n.\n.\n";

	for (int i = size - 10; i < size; i++) {
		cout << result[i] << "\n";
	}
}

void initData(string file, int **data, int *sizeX, int *sizeY) {
	int x;
	int y;
	long long i = 0;
	cout << "Reading "<< file << "... \n";
	ifstream myfile(file);
	if (myfile.is_open()) {
		myfile >> x;
		myfile >> y;

		int *temp = new int[x * y];
		for( i=0; i < x * y; i++){
			myfile >> temp[(int)i];
		}
		myfile.close();
		*data = temp;
		*sizeX = x;
		*sizeY = y;
	}
	else {
		cout << "ERROR: File " << file << " not found!\n";
		exit(0);
	}
	cout << i << " entries imported\n";
}

// Don't change this code
// We will evaluate your correctness based on the saved result, not printed output
void saveResult(string file, int data[], int sizeX, int sizeY) {
	long long i = 0;
	cout << "Saving data to "<< file <<"... \n";
	ofstream myfile(file, std::ofstream::out);
	if (myfile.is_open()) {
		myfile << sizeX << "\n";
		myfile << sizeY << "\n";
		for (i = 0; i < sizeX * sizeY; i++){
			myfile << data[i] << "\n";
		}
		myfile.close();
	}
	else {
		cout << "ERROR: Cannot save to " << file << "!\n";
		exit(0);
	}
	cout << i << " entries saved\n";
}

//TODO: Implement the kernel function

__global__ void sharpen(int *data, int *result, int *FILTER, int sizeX, int sizeY, int FILTER_WIDTH){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(idx < sizeX * sizeY) {
		int fltrOffSetI = (idx / sizeX) - FILTER_WIDTH/2;
		int fltrOffSetJ = (idx % sizeX) - FILTER_WIDTH/2;
		
		int value = 0;
		for (int i = fltrOffSetI; i < fltrOffSetI + FILTER_WIDTH; i++){
			for (int j = fltrOffSetJ; j < fltrOffSetJ + FILTER_WIDTH; j++){
				if (i > -1 && i < sizeY && j > -1 && j < sizeX)	value += FILTER[(i-fltrOffSetI)*FILTER_WIDTH + (j-fltrOffSetJ)] * data[i*sizeX + j];
			}
		}

		if (value < 0) value =  0;
		if (value > 255) value = 255;
		result[idx] = value;
	}
}

// GPU implementation
void GPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image

	// TODO: malloc memory, copy input "from host to device"

	int *d_data, *d_result, *d_FILTER;
	int size = (sizeX * sizeY) * sizeof(int);

	hipMalloc((void **)&d_data, size);
	hipMalloc((void **)&d_result, size);
	hipMalloc((void **)&d_FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int));

	hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
	hipMemcpy(d_FILTER, FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int), hipMemcpyHostToDevice);
	

	// Start timer for kernel
	// Don't change this function
	auto startKernel = chrono::steady_clock::now();
	const int n_blocks = (sizeX * sizeY)/BLOCK_SIZE;

	// TODO: call the kernel function
	sharpen<<<n_blocks, BLOCK_SIZE>>>(d_data, d_result, d_FILTER, sizeX, sizeY, FILTER_WIDTH);
	// End timer for kernel and display kernel time
	hipDeviceSynchronize(); // <- DO NOT REMOVE
	auto endKernel = chrono::steady_clock::now();
	cout << "Kernel Elapsed time: " << chrono::duration <double, milli>(endKernel - startKernel).count() << "ms\n";

	// TODO: copy reuslt from device to host
	hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

	// TODO: free device memory <- important, keep your code clean
	hipFree(d_data); hipFree(d_result); hipFree(d_FILTER);
}


// CPU implementation
void CPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image
	// TODO: sharpen the image with filter
	//       apply zero padding for the border
	long long idx = 0;
	for (idx = 0; idx < sizeX * sizeY; idx++){
		int fltrOffSetI = (idx / sizeX) - FILTER_WIDTH/2;
		int fltrOffSetJ = (idx % sizeX) - FILTER_WIDTH/2;
		
		int value = 0;
		for (int i = fltrOffSetI; i < fltrOffSetI + FILTER_WIDTH; i++){
			for (int j = fltrOffSetJ; j < fltrOffSetJ + FILTER_WIDTH; j++){
				if (i > -1 && i < sizeY && j > -1 && j < sizeX)	value += FILTER[(i-fltrOffSetI)*FILTER_WIDTH + (j-fltrOffSetJ)] * data[i*sizeX + j];
			}
		}

		if (value < 0) value =  0;
		if (value > 255) value = 255;
		result[idx] = value;
	}
}

// The input is a 2D grayscale image
// The image is flattened into a text file of pixel values.
int main(int argc, char *argv[]) {
	string inputFile = (argc == 1) ? "image_grey.txt" : argv[1];

	int sizeX;
	int sizeY;
	int *dataForCPUTest;
	int *dataForGPUTest;	

	initData(inputFile, &dataForCPUTest, &sizeX, &sizeY);
	initData(inputFile, &dataForGPUTest, &sizeX, &sizeY);

	int size = sizeX * sizeY;
	int *resultForCPUTest = new int[size];
	int *resultForGPUTest = new int[size];

	cout << "\n";

	cout << "CPU Implementation\n";

	auto startCPU = chrono::steady_clock::now();
	CPU_Test(dataForCPUTest, resultForCPUTest, sizeX, sizeY);
	auto endCPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endCPU - startCPU).count() << "ms\n";
	// For debug
	// displayResult(dataForCPUTest, resultForCPUTest, size);

	saveResult("grey_result_CPU.txt",resultForCPUTest, sizeX, sizeY);

	cout << "\n";
	cout << "GPU Implementation\n";

	auto startGPU = chrono::steady_clock::now();
	GPU_Test(dataForGPUTest, resultForGPUTest, sizeX, sizeY);
	auto endGPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endGPU - startGPU).count() << "ms\n";

	// For debug
	// displayResult(dataForGPUTest, resultForGPUTest, size);
	saveResult("grey_result_GPU.txt",resultForGPUTest, sizeX, sizeY);

	return 0;
}
